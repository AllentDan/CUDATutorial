
#include <hip/hip_runtime.h>

#include "hipblas.h"
#include "hiprand/hiprand.h"
#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <math.h>


// Check result on the CPU
void verify_result(float *a, float *b, float *c, int N) {
  float temp;
  float epsilon=0.001;
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      temp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        temp += a[k * N + i] * b[j * N + k];
      }

      // Check against the CPU result
      assert(fabs(temp - c[j * N + i] < epsilon));
    }
  }
}

int main(){
    //problem size
    int n = 1<<10;
    size_t bytes = n * n * sizeof(float);

    // Declare pointers to matrices on device and host
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    
    // Allocate memory
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Pseudo random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // set the seed
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    // Fill the matrix with random numbers on the device
    hiprandGenerateUniform(prng, d_a, n*n);
    hiprandGenerateUniform(prng, d_b, n*n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
}
