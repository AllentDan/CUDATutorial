
#include <hip/hip_runtime.h>

#include "hipblas.h"
#include "hiprand/hiprand.h"
#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <math.h>


// Check result on the CPU
void verify_result(float *a, float *b, float *c, int N) {
  float temp;
  float epsilon=0.001;
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      temp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        temp += a[k * N + i] * b[j * N + k];
      }

      // Check against the CPU result
      assert(fabs(temp - c[j * N + i] < epsilon));
    }
  }
}

int main(){
    //problem size
    int n = 1<<10;
    size_t bytes = n * n * sizeof(float);

    // Declare pointers to matrices on device and host
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    
    // Allocate memory
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Pseudo random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // set the seed
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    // Fill the matrix with random numbers on the device
    hiprandGenerateUniform(prng, d_a, n*n);
    hiprandGenerateUniform(prng, d_b, n*n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // scalaing factors
    float alpha = 1.0f;
    float beta = 0.0f;

    // Calculate: c = (alpha*A) * B + (beta*C)
    // (m X n) * (n X k) = (m X k)
    // Signature: handle, operation, operation, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);
    // where operation can be CUBLAS_OP_N or CUBLAS_OP_T, the former do nothing while the latter do transpose for the matrix

    // Copy back the threee matrices
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Verify solution
    verify_result(h_a, h_b, h_c, n);
    
    printf("COMPLETED SUCCESSFULLY\n");

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
